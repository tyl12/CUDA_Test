#include "hip/hip_runtime.h"
#include "funset.hpp"
#include <iostream>
#include <hip/hip_runtime.h> // For the CUDA runtime routines (prefixed with "cuda_")
#include <>
#include "common.hpp"

/* __global__: 函数类型限定符;在设备上运行;在主机端调用,计算能力3.2及以上可以在
设备端调用;声明的函数的返回值必须是void类型;对此类型函数的调用是异步的,即在
设备完全完成它的运行之前就返回了;对此类型函数的调用必须指定执行配置,即用于在
设备上执行函数时的grid和block的维度,以及相关的流(即插入<<<   >>>运算符);
a kernel,表示此函数为内核函数(运行在GPU上的CUDA并行计算函数称为kernel(内核函
数),内核函数必须通过__global__函数类型限定符定义);*/
__global__ static void long_vector_add(const float *A, const float *B, float *C, int elements_num)
{
	/* gridDim: 内置变量,用于描述线程网格的维度,对于所有线程块来说,这个
	变量是一个常数,用来保存线程格每一维的大小,即每个线程格中线程块的数量.
	一个grid为三维,为dim3类型；
	blockDim: 内置变量,用于说明每个block的维度与尺寸.为dim3类型,包含
	了block在三个维度上的尺寸信息;对于所有线程块来说,这个变量是一个常数,
	保存的是线程块中每一维的线程数量;
	blockIdx: 内置变量,变量中包含的值就是当前执行设备代码的线程块的索引;用
	于说明当前thread所在的block在整个grid中的位置,blockIdx.x取值范围是
	[0,gridDim.x-1],blockIdx.y取值范围是[0, gridDim.y-1].为uint3类型,
	包含了一个block在grid中各个维度上的索引信息;
	threadIdx: 内置变量,变量中包含的值就是当前执行设备代码的线程索引;用于
	说明当前thread在block中的位置;如果线程是一维的可获取threadIdx.x,如果
	是二维的还可获取threadIdx.y,如果是三维的还可获取threadIdx.z;为uint3类
	型,包含了一个thread在block中各个维度的索引信息 */
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid == 0) {
		printf("blockDim.x = %d, gridDim.x = %d\n", blockDim.x, gridDim.x); // support pritnf, but don't support fprintf
	}
	while (tid < elements_num) {
		C[tid] = A[tid] * B[tid];
		tid += blockDim.x * gridDim.x;
	}
}

int long_vector_add_gpu(const float* A, const float* B, float* C, int elements_num, float* elapsed_time)
{
	/* hipEvent_t: CUDA event types,结构体类型, CUDA事件,用于测量GPU在某
	个任务上花费的时间,CUDA中的事件本质上是一个GPU时间戳,由于CUDA事件是在
	GPU上实现的,因此它们不适于对同时包含设备代码和主机代码的混合代码计时*/
	hipEvent_t start, stop;
	// hipEventCreate: 创建一个事件对象,异步启动
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// hipEventRecord: 记录一个事件,异步启动,start记录起始时间
	//hipEventRecord(start, 0);

	size_t lengthA{ elements_num * sizeof(float) }, lengthB{ elements_num * sizeof(float) };
	size_t lengthC{ elements_num * sizeof(float) };
	float *d_A{ nullptr }, *d_B{ nullptr }, *d_C{ nullptr };

	// hipMalloc: 在设备端分配内存
	hipMalloc(&d_A, lengthA);
	hipMalloc(&d_B, lengthB);
	hipMalloc(&d_C, lengthC);

	/* hipMemcpy: 在主机端和设备端拷贝数据,此函数第四个参数仅能是下面之一:
	(1). hipMemcpyHostToHost: 拷贝数据从主机端到主机端
	(2). hipMemcpyHostToDevice: 拷贝数据从主机端到设备端
	(3). hipMemcpyDeviceToHost: 拷贝数据从设备端到主机端
	(4). hipMemcpyDeviceToDevice: 拷贝数据从设备端到设备端
	(5). hipMemcpyDefault: 从指针值自动推断拷贝数据方向,需要支持
	统一虚拟寻址(CUDA6.0及以上版本)
	hipMemcpy函数对于主机是同步的 */
	hipMemcpy(d_A, A, lengthA, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, lengthB, hipMemcpyHostToDevice);

	/* <<< >>>: 为CUDA引入的运算符,指定线程网格和线程块维度等,传递执行参
	数给CUDA编译器和运行时系统,用于说明内核函数中的线程数量,以及线程是如何
	组织的;尖括号中这些参数并不是传递给设备代码的参数,而是告诉运行时如何
	启动设备代码,传递给设备代码本身的参数是放在圆括号中传递的,就像标准的函
	数调用一样;不同计算能力的设备对线程的总数和组织方式有不同的约束;必须
	先为kernel中用到的数组或变量分配好足够的空间,再调用kernel函数,否则在
	GPU计算时会发生错误,例如越界等;
	使用运行时API时,需要在调用的内核函数名与参数列表直接以<<<Dg,Db,Ns,S>>>
	的形式设置执行配置,其中：Dg是一个dim3型变量,用于设置grid的维度和各个
	维度上的尺寸.设置好Dg后,grid中将有Dg.x*Dg.y*Dg.z个block;Db是
	一个dim3型变量,用于设置block的维度和各个维度上的尺寸.设置好Db后,每个
	block中将有Db.x*Db.y*Db.z个thread;Ns是一个size_t型变量,指定各块为此调
	用动态分配的共享存储器大小,这些动态分配的存储器可供声明为外部数组
	(extern __shared__)的其他任何变量使用;Ns是一个可选参数,默认值为0;S为
	hipStream_t类型,用于设置与内核函数关联的流.S是一个可选参数,默认值0. */
	//long_vector_add << < 512, 512 >> >(d_A, d_B, d_C, elements_num);

	hipEventRecord(start, 0);

	long_vector_add << < 1024*4, 512 >> >(d_A, d_B, d_C, elements_num);

	hipEventRecord(stop, 0);
	/* hipDeviceSynchronize: kernel的启动是异步的, 为了定位它是否出错, 一
	般需要加上hipDeviceSynchronize函数进行同步; 将会一直处于阻塞状态,直到
	前面所有请求的任务已经被全部执行完毕,如果前面执行的某个任务失败,将会
	返回一个错误；当程序中有多个流,并且流之间在某一点需要通信时,那就必须
	在这一点处加上同步的语句,即hipDeviceSynchronize；异步启动
	reference: https://stackoverflow.com/questions/11888772/when-to-call-cudadevicesynchronize */
	//hipDeviceSynchronize();

	hipMemcpy(C, d_C, lengthA, hipMemcpyDeviceToHost);

	// hipFree: 释放设备上由hipMalloc函数分配的内存
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	// hipEventRecord: 记录一个事件,异步启动,stop记录结束时间
	//hipEventRecord(stop, 0);
	// hipEventSynchronize: 事件同步,等待一个事件完成,异步启动
	hipEventSynchronize(stop);
	// cudaEventElapseTime: 计算两个事件之间经历的时间,单位为毫秒,异步启动
	hipEventElapsedTime(elapsed_time, start, stop);
	// hipEventDestroy: 销毁事件对象,异步启动
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}
