#include "hip/hip_runtime.h"
#include "funset.hpp"
#include <iostream>
#include <algorithm>
#include <memory>
#include <vector>
#include <hip/hip_runtime.h> // For the CUDA runtime routines (prefixed with "cuda_")
#include <>
#include "common.hpp"

/* __global__: 函数类型限定符;在设备上运行;在主机端调用,计算能力3.2及以上可以在
设备端调用;声明的函数的返回值必须是void类型;对此类型函数的调用是异步的,即在
设备完全完成它的运行之前就返回了;对此类型函数的调用必须指定执行配置,即用于在
设备上执行函数时的grid和block的维度,以及相关的流(即插入<<<   >>>运算符);
a kernel,表示此函数为内核函数(运行在GPU上的CUDA并行计算函数称为kernel(内核函
数),内核函数必须通过__global__函数类型限定符定义); */
__global__ static void calculate_histogram(const unsigned char* data, int length, unsigned int* hist)
{
	/* __shared__: 变量类型限定符；使用__shared__限定符，或者与__device__限
	定符连用，此时声明的变量位于block中的共享存储器空间中，与block具有相同
	的生命周期，仅可通过block内的所有线程访问；__shared__和__constant__变量
	默认为是静态存储；在__shared__前可以加extern关键字，但表示的是变量大小
	由执行参数确定；__shared__变量在声明时不能初始化；可以将CUDA C的关键字
	__shared__添加到变量声明中，这将使这个变量驻留在共享内存中；CUDA C编译
	器对共享内存中的变量与普通变量将分别采取不同的处理方式 */
	// clear out the accumulation buffer called temp since we are launched with
	// 256 threads, it is easy to clear that memory with one write per thread
	__shared__  unsigned int temp[256]; // 共享内存缓冲区
	temp[threadIdx.x] = 0;
	/* __syncthreads: 对线程块中的线程进行同步；CUDA架构将确保，除非线程块
	中的每个线程都执行了__syncthreads()，否则没有任何线程能执行
	__syncthreads()之后的指令;在同一个block中的线程通过共享存储器(shared
	memory)交换数据，并通过栅栏同步(可以在kernel函数中需要同步的位置调用
	__syncthreads()函数)保证线程间能够正确地共享数据；使用clock()函数计时，
	在内核函数中要测量的一段代码的开始和结束的位置分别调用一次clock()函数，
	并将结果记录下来。由于调用__syncthreads()函数后，一个block中的所有
	thread需要的时间是相同的，因此只需要记录每个block执行需要的时间就行了，
	而不需要记录每个thread的时间 */
	__syncthreads();

	/* gridDim: 内置变量,用于描述线程网格的维度,对于所有线程块来说,这个
	变量是一个常数,用来保存线程格每一维的大小,即每个线程格中线程块的数量.
	为dim3类型；
	blockDim: 内置变量,用于说明每个block的维度与尺寸.为dim3类型,包含
	了block在三个维度上的尺寸信息;对于所有线程块来说,这个变量是一个常数,
	保存的是线程块中每一维的线程数量;
	blockIdx: 内置变量,变量中包含的值就是当前执行设备代码的线程块的索引;用
	于说明当前thread所在的block在整个grid中的位置,blockIdx.x取值范围是
	[0,gridDim.x-1],blockIdx.y取值范围是[0, gridDim.y-1].为uint3类型,
	包含了一个block在grid中各个维度上的索引信息;
	threadIdx: 内置变量,变量中包含的值就是当前执行设备代码的线程索引;用于
	说明当前thread在block中的位置;如果线程是一维的可获取threadIdx.x,如果
	是二维的还可获取threadIdx.y,如果是三维的还可获取threadIdx.z;为uint3类
	型,包含了一个thread在block中各个维度的索引信息 */
	// calculate the starting index and the offset to the next block that each thread will be processing
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while (i < length) {
		/* atomicAdd: 原子操作,底层硬件将确保当执行这些原子操作时，其
		它任何线程都不会读取或写入地址addr上的值。原子函数(atomic
		function)对位于全局或共享存储器的一个32位或64位字执行
		read-modify-write的原子操作。也就是说，当多个线程同时访问全局或
		共享存储器的同一位置时，保证每个线程能够实现对共享可写数据的互
		斥操作：在一个操作完成之前，其它任何线程都无法访问此地址。之所
		以将这一过程称为原子操作，是因为每个线程的操作都不会影响到其它
		线程。换句话说，原子操作能够保证对一个地址的当前操作完成之前，
		其它线程都不能访问这个地址。
		atomicAdd(addr,y)：将生成一个原子的操作序列，这个操作序列包括读
		取地址addr处的值，将y增加到这个值，以及将结果保存回地址addr。 */
		atomicAdd(&temp[data[i]], 1);
		i += stride;
	}

	// sync the data from the above writes to shared memory then add the shared memory values to the values from
	// the other thread blocks using global memory atomic adds same as before, since we have 256 threads,
	// updating the global histogram is just one write per thread!
	__syncthreads();
	// 将每个线程块的直方图合并为单个最终的直方图
	atomicAdd(&(hist[threadIdx.x]), temp[threadIdx.x]);
}

int calculate_histogram_gpu(const unsigned char* data, int length, unsigned int* hist, unsigned int& value, float* elapsed_time)
{
	/* hipEvent_t: CUDA event types,结构体类型, CUDA事件,用于测量GPU在某
	个任务上花费的时间,CUDA中的事件本质上是一个GPU时间戳,由于CUDA事件是在
	GPU上实现的,因此它们不适于对同时包含设备代码和主机代码的混合代码计时 */
	hipEvent_t start, stop;
	// hipEventCreate: 创建一个事件对象,异步启动
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// hipEventRecord: 记录一个事件,异步启动,start记录起始时间
	hipEventRecord(start, 0);

	unsigned char* dev_buffer{ nullptr };
	unsigned int* dev_hist{ nullptr };

	// hipMalloc: 在设备端分配内存
	hipMalloc(&dev_buffer, length);
	hipMalloc(&dev_hist, 256 * sizeof(unsigned int));

	/* hipMemcpy: 在主机端和设备端拷贝数据,此函数第四个参数仅能是下面之一:
	(1). hipMemcpyHostToHost: 拷贝数据从主机端到主机端
	(2). hipMemcpyHostToDevice: 拷贝数据从主机端到设备端
	(3). hipMemcpyDeviceToHost: 拷贝数据从设备端到主机端
	(4). hipMemcpyDeviceToDevice: 拷贝数据从设备端到设备端
	(5). hipMemcpyDefault: 从指针值自动推断拷贝数据方向,需要支持
	统一虚拟寻址(CUDA6.0及以上版本)
	hipMemcpy函数对于主机是同步的 */
	hipMemcpy(dev_buffer, data, length, hipMemcpyHostToDevice);

	/* hipMemset: 存储器初始化函数,在GPU内存上执行。用指定的值初始化或设置
	设备内存 */
	hipMemset(dev_hist, 0, 256 * sizeof(unsigned int));

	// hipDeviceProp_t: cuda设备属性结构体
	// kernel launch - 2x the number of mps gave best timing
	hipDeviceProp_t prop;
	// hipGetDeviceProperties: 获取GPU设备相关信息
	hipGetDeviceProperties(&prop, 0);
	// hipDeviceProp_t::multiProcessorCount: 设备上多处理器的数量
	int blocks = prop.multiProcessorCount;
	fprintf(stderr, "multiProcessorCount: %d\n", blocks);

	/* <<< >>>: 为CUDA引入的运算符,指定线程网格和线程块维度等,传递执行参
	数给CUDA编译器和运行时系统,用于说明内核函数中的线程数量,以及线程是如何
	组织的;尖括号中这些参数并不是传递给设备代码的参数,而是告诉运行时如何
	启动设备代码,传递给设备代码本身的参数是放在圆括号中传递的,就像标准的函
	数调用一样;不同计算能力的设备对线程的总数和组织方式有不同的约束;必须
	先为kernel中用到的数组或变量分配好足够的空间,再调用kernel函数,否则在
	GPU计算时会发生错误,例如越界等;
	使用运行时API时,需要在调用的内核函数名与参数列表直接以<<<Dg,Db,Ns,S>>>
	的形式设置执行配置,其中：Dg是一个dim3型变量,用于设置grid的维度和各个
	维度上的尺寸.设置好Dg后,grid中将有Dg.x*Dg.y*Dg.z个block;Db是
	一个dim3型变量,用于设置block的维度和各个维度上的尺寸.设置好Db后,每个
	block中将有Db.x*Db.y*Db.z个thread;Ns是一个unsigned int型变量,指定各块为此调
	用动态分配的共享存储器大小,这些动态分配的存储器可供声明为外部数组
	(extern __shared__)的其他任何变量使用;Ns是一个可选参数,默认值为0;S为
	hipStream_t类型,用于设置与内核函数关联的流.S是一个可选参数,默认值0. */
	// 当线程块的数量为GPU中处理器数量的2倍时，将达到最优性能
	calculate_histogram << <blocks * 4, 256 >> >(dev_buffer, length, dev_hist);

	hipMemcpy(hist, dev_hist, 256 * sizeof(unsigned int), hipMemcpyDeviceToHost);

	value = 0;
	for (int i = 0; i < 256; ++i) {
		value += hist[i];
	}

	// hipFree: 释放设备上由hipMalloc函数分配的内存
	hipFree(dev_buffer);
	hipFree(dev_hist);

	// hipEventRecord: 记录一个事件,异步启动,stop记录结束时间
	hipEventRecord(stop, 0);
	// hipEventSynchronize: 事件同步,等待一个事件完成,异步启动
	hipEventSynchronize(stop);
	// cudaEventElapseTime: 计算两个事件之间经历的时间,单位为毫秒,异步启动
	hipEventElapsedTime(elapsed_time, start, stop);
	// hipEventDestroy: 销毁事件对象,异步启动
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}
